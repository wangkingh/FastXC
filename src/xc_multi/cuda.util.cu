#include "cuda.util.cuh"

const float RAMUPPERBOUND = 0.9;

void DimCompute1D(dim3 *pdimgrd, dim3 *pdimblk, size_t width)
{
  // 设置线程块大小，假设 BLOCKX1D 是预定义的每个线程块的线程数
  pdimblk->x = BLOCKX1D;

  // 计算所需的网格大小，确保能够覆盖所有的元素
  pdimgrd->x = (width + BLOCKX1D - 1) / BLOCKX1D;
}

void DimCompute(dim3 *pdimgrd, dim3 *pdimblk, size_t width, size_t height)
{
  pdimblk->x = BLOCKX;
  pdimblk->y = BLOCKY;

  pdimgrd->x = (width + BLOCKX - 1) / BLOCKX;
  pdimgrd->y = (height + BLOCKY - 1) / BLOCKY;
}

int EstimateGpuBatch(size_t gpu_id, int nspec, int nstep, int gpu_task_num)
{
  size_t availram = QueryAvailGpuRam(gpu_id) / gpu_task_num;
  int nfft = 2 * (nspec - 1);

  size_t step = 100;     // 初始步长
  size_t min_step = 1;   // 最小步长
  size_t src_count = 0;  // 源/台 的数量
  size_t pair_count = 0; // 总共涉及多少台站对的计算
  size_t cufftram = 0;
  size_t req_input_ram = 0;
  size_t req_output_ram = 0;
  size_t req_final_output_ram = 0;
  size_t req_final_cc_ram = 0;
  size_t reqram = 0;

  int rank = 1;
  int n[1] = {nfft};
  int inembed[1] = {nfft};
  int onembed[1] = {nfft};
  int istride = 1;
  int idist = nfft;
  int ostride = 1;
  int odist = nfft;
  hipfftType type = HIPFFT_C2R;

  while (true)
  {
    src_count += step;
    pair_count = src_count * src_count;
    req_input_ram = 2 * src_count * nspec * nstep * sizeof(hipComplex);
    req_output_ram = pair_count * nfft * nstep * sizeof(hipComplex);
    req_final_output_ram = pair_count * nfft * sizeof(hipComplex);
    req_final_cc_ram = pair_count * nfft * sizeof(float);
    hipfftEstimateMany(rank, n, inembed, istride, idist, onembed, ostride, odist, type, pair_count, &cufftram);
    reqram = req_input_ram + req_output_ram + req_final_output_ram + req_final_cc_ram + cufftram;

    if (reqram > availram)
    {
      if (step > min_step)
      {
        src_count -= step; // 回退到安全配置
        step /= 2;         // 减小步长
        reqram = 0;        // 重置内存计算，避免使用错误的内存值
      }
      else
      {
        src_count -= step; // 最后尝试步长仍然过大，需要回退
        break;             // 退出循环
      }
    }
  }
  return src_count; // 返回成功
}

size_t QueryAvailGpuRam(size_t deviceID)
{
  size_t freeram, totalram;
  hipSetDevice(deviceID);
  CUDACHECK(hipMemGetInfo(&freeram, &totalram));
  freeram *= RAMUPPERBOUND;

  const size_t gigabytes = 1L << 30;
  printf("Avail gpu ram: %.3f GB\n", freeram * 1.0 / gigabytes);
  return freeram;
}

void CufftPlanAlloc(hipfftHandle *pHandle, int rank, int *n, int *inembed,
                    int istride, int idist, int *onembed, int ostride,
                    int odist, hipfftType type, int batch)
{
  // create cufft plan
  CUFFTCHECK(hipfftPlanMany(pHandle, rank, n, inembed, istride, idist, onembed,
                           ostride, odist, type, batch));
}

void GpuMalloc(void **pptr, size_t sz) { CUDACHECK(hipMalloc(pptr, sz)); }

void GpuCalloc(void **pptr, size_t sz)
{
  CUDACHECK(hipMalloc(pptr, sz));

  CUDACHECK(hipMemset(*pptr, 0, sz));
}

void GpuFree(void **pptr)
{
  CUDACHECK(hipFree(*pptr));
  *pptr = NULL;
}